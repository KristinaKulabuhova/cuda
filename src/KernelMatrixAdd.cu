#include "hip/hip_runtime.h"
#include <KernelMatrixAdd.cuh>

__global__ void KernelMatrixAdd(int height, int width, int pitch, float* A, float* B, float* result) {
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;

	const int stride_x = blockDim.x * gridDim.x;
	const int stride_y = blockDim.y * gridDim.y;

  for (int row = y; row < height; row += stride_y) {
    float* A_y = reinterpret_cast<float*>(reinterpret_cast<char*>(A) + row * pitch);
    float* B_y = reinterpret_cast<float*>(reinterpret_cast<char*>(B) + row * pitch);
    float* res_y = reinterpret_cast<float*>(reinterpret_cast<char*>(result) + row * pitch);

    for(int col = x; col < width; col += stride_x) {
      res_y[col] = A_y[col] + B_y[col];
    }
	}
}

