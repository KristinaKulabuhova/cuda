#include "hip/hip_runtime.h"
#include <MatrixVectorMul.cuh>

__global__
void MatrixVectorMul(int height, int width, float* matrix, float* vector, float* result) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int stride_x = blockDim.x * gridDim.x;
	const int stride_y = blockDim.y * gridDim.y;

    for (int row = y; row < height; row += stride_row) {
        float* A_y = matrix + row * width;
        for(int col = x; col < width; col += stride_col) {
            atomicAdd(&result[row], A_y[col] * vector[col]);
        }
	}
}

