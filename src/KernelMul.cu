#include "hip/hip_runtime.h"
#include <KernelMul.cuh>

__global__ void KernelMul(int numElements, float* x, float* y, float* result) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  
  for (int i = idx; i < numElements; i += stride) {
      result[i] = x[i] + y[i];
  }
}

