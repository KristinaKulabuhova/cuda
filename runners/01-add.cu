#include <cstdint>
#include <iostream>

using std::cout;
using std::endl;

#include "KernelAdd.cuh"


int main() {
  const int N = (1 << 28);
  const int block_size = 256;
  const int n_blocks = (N + block_size - 1) / block_size;
  float *x, *y, *sum;
  const uint64_t array_byte_len = N * sizeof(*x);
  float time;

	hipMallocManaged(&x, array_byte_len);
	hipMallocManaged(&y, array_byte_len);
	hipMallocManaged(&sum, array_byte_len);

	for (int i = 0; i < N; ++i) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}
  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  KernelAdd<<<n_blocks, block_size>>>(numElements, x, y, sum);

  hipEventRecord(stop);
  hipEventSynchronize(stop);
	hipDeviceSynchronize();
  hipEventElapsedTime(&time, start, stop);
  std::cout << time << std::endl;

  for(int i = 0; i < N; ++i) {
    assert(sum[i] == 3.0f);
  }

  hipFree(x);
  hipFree(y);
  hipFree(sum);

  return 0;
}
