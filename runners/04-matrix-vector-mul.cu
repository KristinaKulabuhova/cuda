#include <cassert>
#include <iostream>

#include <MatrixVectorMul.cuh>

const float TOLERANCE = 0.001f;

void FillMatrix(float* mat, int W, int H, float value) {
}

int main() {
  const int W = 10000;
  const int H = 10000;

  dim3 block_size(32, 32);
  dim3 n_blocks((W + block_size.x - 1) / block_size.x, (H + block_size.y - 1) / block_size.y);

  float *h_A = new float[W * H];
  float *h_v = new float[W];
  float *h_result = new float[W];

  for(int row = 0; row < H; ++row) {
    for(int col = 0; col < W; ++col) {
      h_A[row * W + col] = 1.0f;
    }
  }

  for (int row = 0; row < W; ++row) {
    h_v[row] = 1.0f;
  }

  float *d_A, *d_v, *d_result;

  hipMalloc(&d_A, W * H * sizeof(float));
  hipMalloc(&d_v, W * sizeof(float));
  hipMalloc(&d_result, W * sizeof(float));

  hipMemcpy(d_A, h_A, W * H * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_v, h_v, W * sizeof(float), hipMemcpyHostToDevice);

  MatrixVectorMul<<<n_blocks, block_size>>>(H, W, d_A, d_v, d_result);
  hipDeviceSynchronize();

  hipMemcpy(h_result, d_result, W * sizeof(float), hipMemcpyDeviceToHost);

  for (int row = 0; row < W; ++row) {
    assert(h_result[row] - 10000.0f <= 0.01f);
  }

  hipFree(d_A);
  hipFree(d_v);
  hipFree(d_result);

  delete[] h_result;
  delete[] h_v;
  delete[] h_A;

  return 0;
}
