#include "hip/hip_runtime.h"
#include <assert.h>
#include <iostream>

#include <KernelMatrixAdd.cuh>

int main() {
  float *d_A, *d_B, *d_C;
  size_t pitch;
  int W = 10000;
  int H = 10000;

  dim3 block_size{32, 32}; // max threads in block 1024
  dim3 n_blocks{(W + block_size.x - 1) / block_size.x, (H + block_size.y - 1) / block_size.y};

  float *h_A = new float[W * H];
  float *h_B = new float[W * H];
  float *h_C = new float[W * H];

  for(int row = 0; row < H; ++row) {
    for(int col = 0; col < W; ++col) {
      h_A[row * W + col] = 1.0f;
      h_B[row * W + col] = 2.0f;
    }
  }

  hipMallocPitch(&d_A, &pitch, W * sizeof(float), H);
  hipMallocPitch(&d_B, &pitch, W * sizeof(float), H);
  hipMallocPitch(&d_C, &pitch, W * sizeof(float), H);

  hipMemcpy2D(d_A, pitch, h_A, W * sizeof(float), W * sizeof(float), H, hipMemcpyHostToDevice);
  hipMemcpy2D(d_B, pitch, h_B, W * sizeof(float), W * sizeof(float), H, hipMemcpyHostToDevice);

  KernelMatrixAdd<<<n_blocks, block_size>>>(H, W, pitch, d_A, d_B, d_C);
	hipDeviceSynchronize();

  hipMemcpy2D(h_C, W * sizeof(float), d_C, pitch, W * sizeof(float), H, hipMemcpyDeviceToHost);

  for (int row = 0; row < H; ++row) {
    for(int col = 0; col < W; ++col) {
      assert(h_C[row * W + col] == 3.0f);
    }
  }

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  delete[] h_A;
  delete[] h_B;
  delete[] h_C;

  return 0;
}